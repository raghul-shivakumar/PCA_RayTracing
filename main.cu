#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include <stdio.h>
#include<hip/hip_runtime_api.h>

__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0);
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char* argv[]) {

    if(argc < 2){
        printf("Please enter the values of nx, ny, ns, tx and ty\n");
        return 0;
    }
    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int ns = atoi(argv[3]);
    int tx = atoi(argv[4]);
    int ty = atoi(argv[5]);

    float ms = 0;
    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";
    FILE* fp;
    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);
    hipEvent_t start_create_world,stop_create_world,start_render_init,stop_render_init,
                    start_render,stop_render;
    // allocate FB
    vec3 *fb;
    hipError_t err;
    err = hipMallocManaged((void **)&fb, fb_size);
    printf("CUDA malloc managed of Frame Buffers: %s\n",hipGetErrorString(err));

    // allocate random state
    hiprandState *d_rand_state;
    err = hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState));
    printf("CUDA malloc d_rand_state: %s\n",hipGetErrorString(err));
    hiprandState *d_rand_state2;
    err = hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState));
    printf("CUDA malloc d_rand_state2: %s\n",hipGetErrorString(err));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    
    err = hipDeviceSynchronize();
    printf("CUDA device synchronize%s\n",hipGetErrorString(err));

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    err = hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *));
    printf("CUDA d_list: %s\n",hipGetErrorString(err));
    hitable **d_world;
    err = hipMalloc((void **)&d_world, sizeof(hitable *));
    printf("CUDA malloc d_world: %s\n",hipGetErrorString(err));
    camera **d_camera;
    err = hipMalloc((void **)&d_camera, sizeof(camera *));
    printf("CUDA malloc d_camera: %s\n",hipGetErrorString(err));
    hipEventCreate(&start_create_world);
    hipEventCreate(&stop_create_world);
    hipEventRecord(start_create_world);
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    err = hipGetLastError();
    printf("CUDA kernel create_world: %s\n",hipGetErrorString(err));
    err = hipDeviceSynchronize();
    printf("CUDA device synchronize: %s\n",hipGetErrorString(err));
    hipEventRecord(stop_create_world);
    ms = 0;
    hipEventElapsedTime(&ms,start_create_world,stop_create_world);
    printf("The time taken by the function create_world() is: %2.3f ms\n",ms);

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    hipEventCreate(&start_render_init);
    hipEventCreate(&stop_render_init);
    hipEventRecord(start_render_init);   
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    err = hipDeviceSynchronize();
    hipEventRecord(stop_render_init);
    printf("CUDA device synchronize: %s\n",hipGetErrorString(err));
    float ms2 = 0;
    hipEventElapsedTime(&ms2,start_render_init,stop_render_init);
    printf("The time taken by the function render_init() is: %2.3f ms\n",ms2);

    hipEventCreate(&start_render);
    hipEventCreate(&stop_render);
    hipEventRecord(start_render); 
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    err = hipDeviceSynchronize();
    printf("CUDA device synchronize: %s\n",hipGetErrorString(err));
    hipEventRecord(stop_render);
    float ms3 = 0;
    hipEventElapsedTime(&ms3,start_render,stop_render);
    err = hipGetLastError();
    printf("CUDA kernel render: %s\n",hipGetErrorString(err));
    hipEventElapsedTime(&ms3,start_render,stop_render);
    printf("The time taken by the function render() is: %2.3f ms\n",ms3); 
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    //std::cerr << "took " << timer_seconds << " seconds for the render function.\n";

    // Output FB as Image
    fp = fopen("output.ppm","wb");
    fprintf(fp,"P3\n");
    fprintf(fp,"%d %d\n",nx,ny);
    fprintf(fp,"255\n");
    
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            fprintf(fp,"%d %d %d\n",ir,ig,ib);
        }
    }
    fclose(fp);

    // clean up
    err = hipDeviceSynchronize();
    printf("CUDA device synchroize: %s\n",hipGetErrorString(err));
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    err = hipGetLastError();
    printf("CUDA kernel free_world: %s\n",hipGetErrorString(err));
    err = hipFree(d_camera);
    printf("CUDA free d_camera: %s\n",hipGetErrorString(err));
    err = hipFree(d_world);
    printf("CUDA free d_world: %s\n",hipGetErrorString(err));
    err = hipFree(d_list);
    printf("CUDA free d_list: %s\n",hipGetErrorString(err));
    err = hipFree(d_rand_state);
    printf("CUDA free d_rand_state: %s\n",hipGetErrorString(err));
    err = hipFree(fb);
    printf("CUDA free fb: %s\n",hipGetErrorString(err));
}
